
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdint.h>
#include <vector>
// #include <curand.h>
// #include <curand_kernel.h>
// #include <cuda.h>

// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// inline void gpuAssert(cudaError_t code, const char *file, int line, bool
// abort = true) {
//     if (code != cudaSuccess) {
//         fprintf(stderr, "GPUassert: %s %s %d\n", cudaGetErrorString(code),
//         file, line); if (abort) exit(code);
//     }
// }

static int hostTable[(int)1e8];
static const int INF = 1e9;
static const int BLOCK_SIZE = 1024;

static __global__ void dreyfusWagnerFirstStep(const int *distances,
                                              int *dynamicTable,
                                              const int nodes, const int mask) {
    int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x, tmp;
    if (nodeIndex >= nodes)
        return;

    for (int subMask = (mask - 1) & mask; subMask;
         subMask = (subMask - 1) & mask) {
        tmp = dynamicTable[subMask * nodes + nodeIndex] +
              dynamicTable[(mask ^ subMask) * nodes + nodeIndex];
        dynamicTable[mask * nodes + nodeIndex] =
            tmp < dynamicTable[mask * nodes + nodeIndex]
                ? tmp
                : dynamicTable[mask * nodes + nodeIndex];
    }
}

static __global__ void dreyfusWagnerSecondStep(const int *distances,
                                               int *dynamicTable,
                                               const int nodes,
                                               const int mask) {
    int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x, tmp;
    if (nodeIndex >= nodes)
        return;

    for (int node2 = 0; node2 < nodes; node2++) {
        tmp = dynamicTable[mask * nodes + node2] +
              distances[nodeIndex * nodes + node2];
        dynamicTable[mask * nodes + nodeIndex] =
            tmp < dynamicTable[mask * nodes + nodeIndex]
                ? tmp
                : dynamicTable[mask * nodes + nodeIndex];
    }
}

static int *
copyDistancesToDevice(const std::vector<std::vector<int>> &distances) {
    for (int i = 0; i < distances.size(); i++) {
        for (int j = 0; j < distances.size(); j++) {
            hostTable[i * distances.size() + j] = distances[i][j];
        }
    }

    int *cudaDistances;
    hipMalloc(&cudaDistances,
               distances.size() * distances.size() * sizeof(int));

    hipMemcpy(cudaDistances, hostTable,
               distances.size() * distances.size() * sizeof(int),
               hipMemcpyHostToDevice);

    return cudaDistances;
}

static int *
copyDynamicTableToDevice(const std::vector<std::vector<int>> &distances,
                         const std::vector<int> &terminals,
                         const int &fullMask) {
    for (int i = 0; i < (fullMask + 1); i++) {
        for (int j = 0; j < distances.size(); j++) {
            hostTable[i * distances.size() + j] = INF;
        }
    }
    for (int i = 0; i < terminals.size() - 1; i++) {
        hostTable[(1 << i) * distances.size() + terminals[i]] = 0;
    }

    int *cudaDynamicTable;
    hipMalloc(&cudaDynamicTable,
               (fullMask + 1) * distances.size() * sizeof(int));

    hipMemcpy(cudaDynamicTable, hostTable,
               (fullMask + 1) * distances.size() * sizeof(int),
               hipMemcpyHostToDevice);

    return cudaDynamicTable;
}

int dreyfusWagner(const std::vector<std::vector<int>> &distances,
                  const std::vector<int> &terminals) {
    if (terminals.size() <= 1) {
        return 0;
    }
    const int fullMask = (1 << (terminals.size() - 1)) - 1;
    int grid_size_nodes = distances.size() / BLOCK_SIZE + 1;

    int *cudaDistances = copyDistancesToDevice(distances);
    int *cudaDynamicTable =
        copyDynamicTableToDevice(distances, terminals, fullMask);

    for (int mask = 1; mask <= fullMask; mask++) {
        dreyfusWagnerFirstStep<<<grid_size_nodes, BLOCK_SIZE>>>(
            cudaDistances, cudaDynamicTable, distances.size(), mask);
        dreyfusWagnerSecondStep<<<grid_size_nodes, BLOCK_SIZE>>>(
            cudaDistances, cudaDynamicTable, distances.size(), mask);
    }

    int result;
    hipMemcpy(&result,
               cudaDynamicTable +
                   (fullMask * distances.size() + terminals.back()),
               1 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cudaDistances);
    hipFree(cudaDynamicTable);

    return result;
}
