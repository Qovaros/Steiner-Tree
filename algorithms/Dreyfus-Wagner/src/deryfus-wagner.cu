#include "hip/hip_runtime.h"
#include "common.h"
#include "dreyfus-wagner.h"
#include <chrono>
#include <iostream>
#include <stdint.h>
#include <vector>

static int hostTable[(int)1e8];
static const int INF = 1e9;
static const int MAX_THREADS = 1024;

static __global__ void dreyfusWagnerFirstStep(const int *distances,
                                              int *dynamicTable, int *masks,
                                              const int nodes,
                                              const int masksStart,
                                              const int masksEnd) {
    int maskIndex = threadIdx.x, nodeIndex = blockIdx.x, tmp;
    if (maskIndex >= masksEnd - masksStart)
        return;
    int mask = masks[masksStart + maskIndex];
    for (int subMask = (mask - 1) & mask; subMask;
         subMask = (subMask - 1) & mask) {
        tmp = dynamicTable[subMask * nodes + nodeIndex] +
              dynamicTable[(mask ^ subMask) * nodes + nodeIndex];
        dynamicTable[mask * nodes + nodeIndex] =
            tmp < dynamicTable[mask * nodes + nodeIndex]
                ? tmp
                : dynamicTable[mask * nodes + nodeIndex];
    }
}

static __global__ void dreyfusWagnerSecondStep(const int *distances,
                                               int *dynamicTable, int *masks,
                                               const int nodes,
                                               const int masksStart,
                                               const int masksEnd) {
    int maskIndex = threadIdx.x, nodeIndex = blockIdx.x, tmp;
    if (maskIndex >= masksEnd - masksStart)
        return;
    int mask = masks[masksStart + maskIndex];
        for (int node2 = 0; node2 < nodes; node2++) {
            tmp = dynamicTable[mask * nodes + node2] +
                  distances[nodeIndex * nodes + node2];
            dynamicTable[mask * nodes + nodeIndex] =
                tmp < dynamicTable[mask * nodes + nodeIndex]
                    ? tmp
                    : dynamicTable[mask * nodes + nodeIndex];
        }
}

static int *
copyDistancesToDevice(const std::vector<std::vector<int>> &distances) {
    for (int i = 0; i < distances.size(); i++) {
        for (int j = 0; j < distances.size(); j++) {
            hostTable[i * distances.size() + j] = distances[i][j];
        }
    }

    int *cudaDistances;
    hipMalloc(&cudaDistances,
               distances.size() * distances.size() * sizeof(int));

    hipMemcpy(cudaDistances, hostTable,
               distances.size() * distances.size() * sizeof(int),
               hipMemcpyHostToDevice);

    return cudaDistances;
}

static int *
copyDynamicTableToDevice(const std::vector<std::vector<int>> &distances,
                         const std::vector<int> &terminals,
                         const int &fullMask) {
    for (int i = 0; i < (fullMask + 1); i++) {
        for (int j = 0; j < distances.size(); j++) {
            hostTable[i * distances.size() + j] = INF;
        }
    }
    for (int i = 0; i < terminals.size() - 1; i++) {
        hostTable[(1 << i) * distances.size() + terminals[i]] = 0;
    }

    int *cudaDynamicTable;
    hipMalloc(&cudaDynamicTable,
               (fullMask + 1) * distances.size() * sizeof(int));

    hipMemcpy(cudaDynamicTable, hostTable,
               (fullMask + 1) * distances.size() * sizeof(int),
               hipMemcpyHostToDevice);

    return cudaDynamicTable;
}

static int *copyMasksToDevice(const int &terminals, const int &fullMask,
                              std::vector<int> &masksBeginings) {
    std::vector<std::vector<int>> masks(terminals);

    for (int mask = 1; mask <= fullMask; mask++) {
        masks[__builtin_popcount(mask) - 1].push_back(mask);
    }

    int *cudaMasksTable;
    hipMalloc(&cudaMasksTable, (fullMask + 1) * sizeof(int));
    for (int i = 0, j = 0; i < masks.size(); j += masks[i].size(), i++) {
        masksBeginings.push_back(j);
        hipMemcpy(cudaMasksTable + j, &masks[i][0],
                   masks[i].size() * sizeof(int), hipMemcpyHostToDevice);
    }
    masksBeginings.push_back(
        masksBeginings[masksBeginings.size() - 1] +
        masks[masks.size() - 1].size()); // could be just +1

    return cudaMasksTable;
}

DreyfusWagnerStatistics
dreyfusWagner(std::vector<std::vector<int>> &distances,
              const std::vector<std::vector<std::pair<int, int>>> &graph,
              const std::vector<int> &terminals) {
    hipFree(NULL);
    DreyfusWagnerStatistics statistics = {0, 0, 0, 0, 0};
    if (terminals.size() <= 1) {
        return statistics;
    }
    const int fullMask = (1 << (terminals.size() - 1)) - 1;
    std::vector<int> masksBeginings;

    auto beforeFloydWarshall = std::chrono::steady_clock::now();
    compouteDistances(distances, graph);
    auto afterFloydWarshall = std::chrono::steady_clock::now();
    statistics.distancesDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(
            afterFloydWarshall - beforeFloydWarshall)
            .count();

    auto beforeCopy = std::chrono::steady_clock::now();
    int *cudaDistances = copyDistancesToDevice(distances);
    int *cudaDynamicTable =
        copyDynamicTableToDevice(distances, terminals, fullMask);
    int *cudaMasksTable =
        copyMasksToDevice(terminals.size() - 1, fullMask, masksBeginings);
    auto afterCopy = std::chrono::steady_clock::now();
    statistics.copyDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(afterCopy -
                                                              beforeCopy)
            .count();
    for (int maskSize = 1; maskSize < masksBeginings.size(); maskSize++) {
        for (int i = 0, block_size;
             i < masksBeginings[maskSize] - masksBeginings[maskSize - 1];
             i += MAX_THREADS) {
            block_size = std::min(
                (masksBeginings[maskSize] - masksBeginings[maskSize - 1] - i),
                MAX_THREADS);
            dreyfusWagnerFirstStep<<<distances.size(), block_size>>>(
                cudaDistances, cudaDynamicTable, cudaMasksTable,
                distances.size(), masksBeginings[maskSize - 1] + i,
                masksBeginings[maskSize] + i);
            dreyfusWagnerSecondStep<<<distances.size(), block_size>>>(
                cudaDistances, cudaDynamicTable, cudaMasksTable,
                distances.size(), masksBeginings[maskSize - 1] + i,
                masksBeginings[maskSize] + i);
        }
    }
    hipMemcpy(&statistics.result,
               cudaDynamicTable +
                   (fullMask * distances.size() + terminals.back()),
               1 * sizeof(int), hipMemcpyDeviceToHost);
    auto end = std::chrono::steady_clock::now();
    statistics.dreyfusWagnerDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(end - afterCopy)
            .count();
    statistics.everythingDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(
            end - beforeFloydWarshall)
            .count();
    hipFree(cudaDistances);
    hipFree(cudaDynamicTable);
    hipFree(cudaMasksTable);
    return statistics;
}
