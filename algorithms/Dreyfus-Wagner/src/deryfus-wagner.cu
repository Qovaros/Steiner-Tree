#include "hip/hip_runtime.h"
#include "common.h"
#include "dreyfus-wagner.h"
#include <chrono>
#include <iostream>
#include <stdint.h>
#include <vector>
// #include <hiprand.h>
// #include <hiprand/hiprand_kernel.h>
// #include <hip/hip_runtime.h>

// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// inline void gpuAssert(hipError_t code, const char *file, int line, bool
// abort = true) {
//     if (code != hipSuccess) {
//         fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
//         file, line); if (abort) exit(code);
//     }
// }

static int hostTable[(int)1e8];
static const int INF = 1e9;
static const int BLOCK_NUMBER = 10;

static __global__ void dreyfusWagnerFirstStep(const int *distances,
                                              int *dynamicTable,
                                              const int nodes, const int mask) {
    int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x, tmp;
    if (nodeIndex >= nodes)
        return;

    for (int subMask = (mask - 1) & mask; subMask;
         subMask = (subMask - 1) & mask) {
        tmp = dynamicTable[subMask * nodes + nodeIndex] +
              dynamicTable[(mask ^ subMask) * nodes + nodeIndex];
        dynamicTable[mask * nodes + nodeIndex] =
            tmp < dynamicTable[mask * nodes + nodeIndex]
                ? tmp
                : dynamicTable[mask * nodes + nodeIndex];
    }
}

static __global__ void dreyfusWagnerSecondStep(const int *distances,
                                               int *dynamicTable,
                                               const int nodes,
                                               const int mask) {
    int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x, tmp;
    if (nodeIndex >= nodes)
        return;

    for (int node2 = 0; node2 < nodes; node2++) {
        tmp = dynamicTable[mask * nodes + node2] +
              distances[nodeIndex * nodes + node2];
        dynamicTable[mask * nodes + nodeIndex] =
            tmp < dynamicTable[mask * nodes + nodeIndex]
                ? tmp
                : dynamicTable[mask * nodes + nodeIndex];
    }
}

static int *
copyDistancesToDevice(const std::vector<std::vector<int>> &distances) {
    for (int i = 0; i < distances.size(); i++) {
        for (int j = 0; j < distances.size(); j++) {
            hostTable[i * distances.size() + j] = distances[i][j];
        }
    }

    int *cudaDistances;
    hipMalloc(&cudaDistances,
               distances.size() * distances.size() * sizeof(int));

    hipMemcpy(cudaDistances, hostTable,
               distances.size() * distances.size() * sizeof(int),
               hipMemcpyHostToDevice);

    return cudaDistances;
}

static __global__ void fillDynamicTable(const int *distances, int *dynamicTable,
                                        const int nodes, const int mask) {
    int nodeIndex = blockDim.x * blockIdx.x + threadIdx.x, tmp;
    if (nodeIndex >= nodes)
        return;

    for (int node2 = 0; node2 < nodes; node2++) {
        tmp = dynamicTable[mask * nodes + node2] +
              distances[nodeIndex * nodes + node2];
        dynamicTable[mask * nodes + nodeIndex] =
            tmp < dynamicTable[mask * nodes + nodeIndex]
                ? tmp
                : dynamicTable[mask * nodes + nodeIndex];
    }
}

static int *
copyDynamicTableToDevice(const std::vector<std::vector<int>> &distances,
                         const std::vector<int> &terminals,
                         const int &fullMask) {
    for (int i = 0; i < (fullMask + 1); i++) {
        for (int j = 0; j < distances.size(); j++) {
            hostTable[i * distances.size() + j] = INF;
        }
    }
    for (int i = 0; i < terminals.size() - 1; i++) {
        hostTable[(1 << i) * distances.size() + terminals[i]] = 0;
    }

    int *cudaDynamicTable;
    hipMalloc(&cudaDynamicTable,
               (fullMask + 1) * distances.size() * sizeof(int));

    hipMemcpy(cudaDynamicTable, hostTable,
               (fullMask + 1) * distances.size() * sizeof(int),
               hipMemcpyHostToDevice);

    return cudaDynamicTable;
}

DreyfusWagnerStatistics
dreyfusWagner(std::vector<std::vector<int>> &distances,
              const std::vector<std::vector<std::pair<int, int>>> &graph,
              const std::vector<int> &terminals) {
    hipFree(NULL);
    DreyfusWagnerStatistics statistics = {0, 0, 0, 0, 0};
    if (terminals.size() <= 1) {
        return statistics;
    }
    const int fullMask = (1 << (terminals.size() - 1)) - 1;
    int block_size = distances.size() / BLOCK_NUMBER + 1;

    auto beforeFloydWarshall = std::chrono::steady_clock::now();
    compouteDistances(distances, graph);
    auto afterFloydWarshall = std::chrono::steady_clock::now();
    statistics.distancesDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(
            afterFloydWarshall - beforeFloydWarshall)
            .count();

    auto beforeCopy = std::chrono::steady_clock::now();
    int *cudaDistances = copyDistancesToDevice(distances);
    int *cudaDynamicTable =
        copyDynamicTableToDevice(distances, terminals, fullMask);
    auto afterCopy = std::chrono::steady_clock::now();
    statistics.copyDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(afterCopy -
                                                              beforeCopy)
            .count();
    for (int mask = 1; mask <= fullMask; mask++) {
        dreyfusWagnerFirstStep<<<BLOCK_NUMBER, block_size>>>(
            cudaDistances, cudaDynamicTable, distances.size(), mask);
        dreyfusWagnerSecondStep<<<BLOCK_NUMBER, block_size>>>(
            cudaDistances, cudaDynamicTable, distances.size(), mask);
    }
    auto end = std::chrono::steady_clock::now();
    statistics.dreyfusWagnerDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(end - afterCopy)
            .count();
    statistics.everythingDuration =
        std::chrono::duration_cast<std::chrono::milliseconds>(
            end - beforeFloydWarshall)
            .count();
    hipMemcpy(&statistics.result,
               cudaDynamicTable +
                   (fullMask * distances.size() + terminals.back()),
               1 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cudaDistances);
    hipFree(cudaDynamicTable);

    return statistics;
}
